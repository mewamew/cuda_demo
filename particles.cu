#include "hip/hip_runtime.h"
﻿#include "particles.h"
#include <hip/hip_runtime.h>
#include <>
#include <vector>
#include <random>
#include <cmath>
#include <cstdio>

// CUDA 设备变量
Particle* d_particles = nullptr;

// CUDA 错误检查宏
#define cudaCheckError(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char* file, int line, bool abort=true)
{
    if (code != hipSuccess) 
    {
        fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

__device__ float distance(float x1, float y1, float x2, float y2) {
    float dx = x2 - x1;
    float dy = y2 - y1;
    return sqrtf(dx*dx + dy*dy);
}

__global__
void CollideParticlesKernel(Particle* d_particles, int count, float dt) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < count) {
        Particle p1 = d_particles[idx];
        
        // 更新碰撞时间
        if (p1.collision_time > 0) {
            p1.collision_time -= dt;
            if (p1.collision_time <= 0) {
                // 恢复为黄色
                p1.color[0] = 1.0f;
                p1.color[1] = 1.0f;
                p1.color[2] = 0.0f;
                p1.collision_time = 0;
            }
        }

        for (int j = idx + 1; j < count; j++) {
            Particle p2 = d_particles[j];
            
            float dist = distance(p1.x, p1.y, p2.x, p2.y);
            if (dist < d_PARTICLE_SIZE * 2) {
                // 碰撞处理代码...

                // 将碰撞的粒子变为红色
                p1.color[0] = 1.0f;  // R
                p1.color[1] = 0.0f;  // G
                p1.color[2] = 0.0f;  // B
                p1.collision_time = 0.1f;  // 设置碰撞时间为1秒

                p2.color[0] = 1.0f;  // R
                p2.color[1] = 0.0f;  // G
                p2.color[2] = 0.0f;  // B
                p2.collision_time = 0.1f;  // 设置碰撞时间为1秒

                d_particles[j] = p2;
            }
        }
        
        d_particles[idx] = p1;
    }
}

// CUDA 核函数，用于更新粒子位置
__global__
void UpdateParticlesKernel(Particle* d_particles, int count) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < count) {
        Particle p = d_particles[idx];
        
        // 更新位置
        p.x += p.vx;
        p.y += p.vy;
        
        // 边界检测和反弹
        if (p.x <= 0 || p.x >= WINDOW_WIDTH) {
            p.vx = -p.vx; 
            p.x = (p.x <= 0) ? 0 : WINDOW_WIDTH;
        }
        if (p.y <= 0 || p.y >= WINDOW_HEIGHT) {
            p.vy = -p.vy;
            p.y = (p.y <= 0) ? 0 : WINDOW_HEIGHT;
        }
        
        d_particles[idx] = p;
    }
}

// 主机函数：初始化粒子
extern "C" void InitializeParticles(Particle* h_particles, int count) {
    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_real_distribution<> dis_angle(-1.0, 1.0);
    std::uniform_real_distribution<> dis_speed(-2.0, 2.0);
    std::uniform_real_distribution<> dis_mass(MIN_MASS, MAX_MASS);  // 质量范围从0.5到2.0

    const int rows = ROWS;
    const int cols = COLS;
    const float spacing = PARTICLE_SIZE * 2; // 空隙大小为粒子大小
    const float startX = (WINDOW_WIDTH - (cols - 1) * spacing) / 2; // 假设窗口宽度为800
    const float startY = (WINDOW_HEIGHT - (rows - 1) * spacing) / 2; // 假设窗口高度为600

    // 设置设备常量
    float h_PARTICLE_SIZE = 1.0f;
    hipMemcpyToSymbol(HIP_SYMBOL(d_PARTICLE_SIZE), &h_PARTICLE_SIZE, sizeof(float));


    for (int i = 0; i < count; ++i) {
        int row = i / cols;
        int col = i % cols;
        
        Particle p;
        p.x = startX + col * spacing;
        p.y = startY + row * spacing;
        
        float angle = static_cast<float>(dis_angle(gen) * 2 * 3.14159);
        float speed = 2.0f + static_cast<float>(dis_speed(gen));
        p.vx = std::cos(angle) * speed;
        p.vy = std::sin(angle) * speed;
        p.mass = static_cast<float>(dis_mass(gen));
        p.color[0] = 1.0f;  // R
        p.color[1] = 1.0f;  // G
        p.color[2] = 0.0f;  // B
        p.collision_time = 0.0f;  // 初始化碰撞时间为0
        h_particles[i] = p;
    }

    // 将数据复制到设备
    cudaCheckError(hipMemcpy(d_particles, h_particles, count * sizeof(Particle), hipMemcpyHostToDevice));
}

// 主机函数：更新粒子
extern "C" void UpdateParticles(int count, Particle* h_particles, float dt) {
    // 调用CUDA核函数
    int threadsPerBlock = 256;
    int blocksPerGrid = (count + threadsPerBlock - 1) / threadsPerBlock;
    
    UpdateParticlesKernel<<<blocksPerGrid, threadsPerBlock>>>(d_particles, count);
    CollideParticlesKernel<<<blocksPerGrid, threadsPerBlock>>>(d_particles, count, dt);
    
    // 将更新后的粒子数据复制回主机
    hipMemcpy(h_particles, d_particles, count * sizeof(Particle), hipMemcpyDeviceToHost);
}

// 主机函数：分配设备内存
extern "C" void AllocateDeviceMemory(int count) {
    cudaCheckError(hipMalloc((void**)&d_particles, count * sizeof(Particle)));
}

// 主机函数：释放设备内存
extern "C" void CleanupCUDA() {
    if (d_particles != nullptr) {
        hipFree(d_particles);
        d_particles = nullptr;
    }
}